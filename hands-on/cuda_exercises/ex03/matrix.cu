
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0


// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);
// Part 2 of 4: implement the kernel
__global__ void kernel( int *a, int dimx, int dimy ) 
{
  dim3 index;
  index.x = threadIdx.x + blockIdx.x * blockDim.x;
//  printf("index.x %d threadIdx.x %d blockIdx.x %d blockDim.x %d \n",index.x,threadIdx.x,blockIdx.x,blockDim.x);
  index.y = threadIdx.y + blockIdx.y * blockDim.y;
//  printf("index.y %d threadIdx.y %d blockIdx.y %d blockDim.y %d \n",index.y,threadIdx.y,blockIdx.y,blockDim.y);

  if (index.x < dimx && index.y < dimy ) {
    int i = index.x + index.y * dimx;
//    printf("i %d x %d y %d dimx %d \n",i,index.x,index.y,dimx);
    a[i] = i;
  }

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char* argv[]) {
    hipSetDevice(MYDEVICE);
// Part 1 and 4 of 4: set the dimensions of the matrix
    int const dimx = (argc > 4) ? std::atoi(argv[1]) : 4;
    int const dimy = (argc > 4) ? std::atoi(argv[2]) : 4;
    std::cout << "MATRIX " << dimx << "x" << dimy << std::endl;
    //    int dimx = 4;
    //    int dimy = 4;
    int num_bytes = dimx*dimy*sizeof(dimx);

    int *d_a=0, *h_a=0; // device and host pointers

    //    h_a = (int*)malloc(num_bytes);
    hipHostMalloc( &h_a,num_bytes , hipHostMallocDefault);
    //allocate memory on the device
    hipMalloc( &d_a,num_bytes );

    if( NULL==h_a || NULL==d_a ) {
        std::cerr << "couldn't allocate memory" << std::endl;
        return 1;
    }

    // Part 2 of 4: define grid and block size and launch the kernel
    dim3 grid, block;
    //    block.x = 2;
    //    block.y = 2;
    block.x = (argc > 4) ? std::atoi(argv[3]) : 2;
    block.y = (argc > 4) ? std::atoi(argv[4]) : 2;
    std::cout << "#THREADS " << block.x << "x" << block.y << std::endl;
    grid.x  = (dimx + block.x - 1)/block.x;
    grid.y  = (dimy + block.y - 1)/block.y;
    std::cout << "#BLOCKS " << grid.x << "x" << grid.y << std::endl;
    

    kernel<<<grid, block>>>( d_a, dimx, dimy );
    // block until the device has completed
    hipDeviceSynchronize();
    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    // device to host copy
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
    // verify the data returned to the host is correct
    for(int row=0; row<dimy; row++)
    {
        for(int col=0; col<dimx; col++)
            assert(h_a[row * dimx + col] == row * dimx + col);
    }
    // free host memory
    //    free( h_a );
    hipHostFree(h_a);

    // free device memory
    hipFree( d_a );

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
